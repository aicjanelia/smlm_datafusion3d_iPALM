#include "hip/hip_runtime.h"
#include <stdio.h>

/*
    expdist   Computes the bhattacharya cost function for two given point set

    SYNOPSIS:
    D = expdist(A, B, m, n, dim, scale_A, scale_B);

    INPUT
        A
            The first particle containing the list of coordinates
        B
            The second particle containing the list of coordinates
        m
            Size of the particle A
        n
            Size of the particle B
        dim
            particles dimension (2D or 3D) 
        scale_A
            uncertainties of particle A 
        scale_B
            uncertainties of particle B 

    OUTPUT
        result
            The distance between particle A and B

    (C) Copyright 2017              Quantitative Imaging Group
        All rights reserved         Faculty of Applied Physics
                                    Delft University of Technology
                                    Lorentzweg 1
                                    2628 CJ Delft
                                    The Netherlands
    Hamidreza Heydarian and Ben van Werkhoven, Feb 2017
*/
#define SQR(X)  ((X)*(X))

#include <math.h>
/* #include <stdio.h> */


#include "expdist_functions.cu"





#ifdef WIN32
__declspec( dllexport )
#endif
double expdist(const double *A, const double *B, int m, int n, int dim, const double *scale_A, const double *scale_B)
{
    int i,j,d;
    int id, jd;
    double dist_ij, cross_term = 0;

    for (i=0;i<m;++i)
    {
        for (j=0;j<n;++j)
        {
            dist_ij = 0;
            for (d=0;d<dim;++d)
            {
                id = i + d * m;
                jd = j + d * n;
                dist_ij = dist_ij + SQR( A[id] - B[jd]);
            }
            cross_term += exp(-dist_ij/(scale_A[i] + scale_B[j]));
        }
    }

    return cross_term;
}




/*
 * The following function is a full 3D implementation of the Bhattacharya distance
 * scale_A is an array with 2 values per localization
 * scale_B contains the pre-rotated matrix of uncertainties for B
 */
template <typename T>
T expdist3D(const T *A, const T *B, const int m, const int n, const T *scale_A, const T *scale_B) {
    int i,j;
    T cross_term = 0.0;
    const int dim = 3;

    T pA[dim];
    T pB[dim];

    for (i=0; i<m; i++) {

        //prefetch point Ai
        for (int d=0; d<dim; d++) {
            int id = i + d * m;
            pA[d] = A[id];
        }

        //assume sigma in x and y are equal and scale_A only stores 2 values per localization
        T Sigma_i[9];
        zero_matrix(Sigma_i);
        Sigma_i[0] = scale_A[i*2+0];   // 0 1 2
        Sigma_i[4] = scale_A[i*2+0];   // 3 4 5
        Sigma_i[8] = scale_A[i*2+1];   // 6 7 8

        for (j=0; j<n; j++) {

            //prefetch point Bj
            for (int d=0; d<dim; d++) {
                int jd = j + d * n;
                pB[d] = B[jd];
            }

            //assume sigma_j has been rotated properly beforehand so that it can be used directly
            T Sigma_j[9];
            load_matrix(Sigma_j, scale_B, j);

            cross_term += compute_expdist_3D<T, 3>(pA, pB, Sigma_i, Sigma_j);
        }
    }

    return cross_term;
}


/*
 * This function rotates the uncertainties for the 3D bhattacharya distance
 *
 * It is assumed that the scale_B array contains 2 uncertainty values per localization in
 * the particle. One value for the uncertainty in X and Y and one for the uncertainty in Z (depth).
 *
 * The output array rotated_scales contains a 3x3 matrix for each localization.
 */
template <typename T>
void rotate_scales(T *rotated_scales, const T *rotation_matrix, const int n, const T *scale_B) {

    T transposed_rotation_matrix[9];
    transpose_matrix<T, 9, 3>(transposed_rotation_matrix, reinterpret_cast<const T(&)[9]>(*rotation_matrix));

    for (int i=0; i<n; i++) {

        rotate_scale(rotated_scales, rotation_matrix, transposed_rotation_matrix, i, scale_B);
    }

}

/*
 * This function rotates the coordinates of the localizations in the B particle for the 3D bhattacharya distance
 *
 * The output array rotated_B contains the x,y,z coordinates of each localization.
 */
template <typename T>
void rotate_B(T *rotated_B, const T *rotation_matrix, const int n, const T *B) {

    for (int i=0; i<n; i++) {
        rotate_B_point(rotated_B, rotation_matrix, i, B);
    }

}




template double expdist3D<double>(const double *A, const double *B, const int m, const int n, const double *scale_A, const double *scale_B);

template void rotate_scales<double>(double *rotated_scales, const double *rotation_matrix, const int n, const double *scale_B);
template void rotate_B<double>(double *rotated_B, const double *rotation_matrix, const int n, const double *B);





